#include "hip/hip_runtime.h"
#include "cuda/mcubes/demos/demo_utils.cuh"
#include "cuda/mcubes/vegasT.cuh"

class F_4_5D {
  public:
    __host__ __device__ double
    operator()(double x, double y, double z, double w, double v)
    {	
	  //return x + y + z + w + v;
      double beta = .5;
      return exp(
        -1.0 * (pow(25., 2.) * pow(x - beta, 2.) + pow(25., 2.) * pow(y - beta, 2.) +
                pow(25., 2.) * pow(z - beta, 2.) + pow(25., 2.) * pow(w - beta, 2.) +
                pow(25., 2.) * pow(v - beta, 2.)));
    }
};

int
main(int argc, char** argv)
{
  int num_repeats = argc > 1 ? std::stoi(argv[1]) : 100;
  double epsrel = 1e-3;
  constexpr int ndim = 5;

  double ncall = 1.0e8;
  int titer = 1;
  int itmax = 1;
  int skip = 0;
  VegasParams params(ncall, titer, itmax, skip);
  double true_value = 1.79132603674879e-06;
  
  double lows[] = {0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  
  F_4_5D integrand;
  std::array<double, 4> required_ncall = {1.e8, 1.e9, 2.e9, 3.e9};
   
  size_t run = 0;
  
  for(auto num_samples : required_ncall){
    params.ncall = num_samples;
    
	signle_invocation_time_and_call<F_4_5D, ndim>(
        integrand, epsrel, true_value, "f4, 5", params, &volume, num_repeats);
	run++;
	if(run > required_ncall.size())
		break;
  }

  return 0;
}