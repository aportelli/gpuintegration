#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"
#include "common/cuda/integrands.cuh"

int
main(int argc, char** argv)
{
  int num_repeats = argc > 1 ? std::stoi(argv[1]) : 11;
  constexpr int ndim = 6;
  F_2_6D integrand;
  quad::Volume<double, ndim> vol;
  call_cubature_rules<F_2_6D, ndim>(integrand, vol, num_repeats);
  return 0;
}
