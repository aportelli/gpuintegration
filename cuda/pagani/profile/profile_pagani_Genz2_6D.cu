#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"
#include "common/cuda/integrands.cuh"

int
main(int argc, char** argv)
{
  int num_repeats = argc > 1 ? std::stoi(argv[1]) : 11;
  constexpr int ndim = 8;
  F_2_8D integrand;
  quad::Volume<double, ndim> vol;
  call_cubature_rules<F_2_8D, ndim>(integrand, vol, num_repeats);
  return 0;
}
