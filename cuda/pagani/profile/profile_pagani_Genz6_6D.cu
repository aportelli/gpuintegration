#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"
#include "common/cuda/integrands.cuh"

int
main(int argc, char** argv)
{
  int num_repeats = argc > 1 ? std::stoi(argv[1]) : 100;
  constexpr int ndim = 6;
  F_6_6D integrand;
  quad::Volume<double, ndim> vol;
  call_cubature_rules<F_6_6D, ndim>(integrand, vol, num_repeats);
  return 0;
}
