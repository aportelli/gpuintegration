#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"
#include "common/cuda/integrands.cuh"

template <typename F,
          int ndim,
          bool use_custom = false,
          int debug = 0,
          int runs_per_esprel = 10>
void
czero_time_and_call(std::string id,
                  double epsrel,
                  std::ostream& outfile,
                  quad::Volume<double, ndim>& vol)
{
	std::vector<double> sharpness_params = {8., 9., 10., 11., 12.};
  for(auto sharpness : sharpness_params){
    using MilliSeconds =
      std::chrono::duration<double, std::chrono::milliseconds::period>;
    double constexpr epsabs = 1.0e-40;
    bool relerr_classification = true;
    Workspace<double, ndim, debug, use_custom> workspace;
    F integrand;
    integrand.sharpness = sharpness;
    integrand.set_true_value();
    auto print_custom = [=](bool use_custom_flag) {
      std::string to_print = use_custom_flag == true ? "custom" : "library";
      return to_print;
    };

    for (int i = 0; i < runs_per_esprel; i++) {
      auto const t0 = std::chrono::high_resolution_clock::now();
      size_t partitions_per_axis = get_partitions_per_axis(ndim);
      Sub_regions<double, ndim> sub_regions(partitions_per_axis);
      constexpr bool predict_split = false;
      constexpr bool collect_iters = false;

      numint::integration_result result =
        workspace.template integrate<F, predict_split, collect_iters>(
          integrand, sub_regions, epsrel, epsabs, vol, relerr_classification);
      MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
      double const absolute_error =
        std::abs(result.estimate - integrand.true_value);

      std::cout.precision(17);
      /*if (i != 0)*/ {
        std::cout << id << "," << ndim << "," << vol.lows[0] << ","
                  << vol.highs[0] << "," 
                  << sharpness << "," 
                  << print_custom(use_custom) << ","
                  << std::fixed << std::scientific << integrand.true_value << ","
                  << epsrel << "," << epsabs << "," << result.estimate << ","
                  << result.errorest << "," << result.nregions << ","
                  << result.status << "," << dt.count() << std::endl;

        outfile << id << "," 
                << ndim << "," << std::scientific 
                << vol.lows[0] << "," << vol.highs[0] << "," 
                << sharpness << "," 
                << print_custom(use_custom) << "," 
                << integrand.true_value << ","
                << epsrel << ","
                << epsabs << "," 
                << result.estimate << "," 
                << result.errorest << "," 
                << result.nregions << "," 
                << result.iters << "," 
                << result.status << ","
                << dt.count() << std::endl;
      }
    }
  }
}

// separable integrands

// semi-separable integrands

// fully separable integrands

int
main()
{
  constexpr bool use_custom = false;
  constexpr int debug = 0;
  constexpr int num_runs = 10;
  std::vector<double> epsrels = {1.e-3, 1.e-4, 1.e-5, 1.e-6, 1.e-7, 1.e-8, 1.e-9};
  std::vector<std::pair<double, double>> volumes = {{0, 1}, {0, 2}};
  std::ofstream outfile("cuda_pagani_czero.csv");
  outfile << "id, ndim, low, high, sharpness, use_custom, true_value, epsrel, epsabs, "
             "estimate, errorest, nregions, completed_iters, status, time"
          << std::endl;
  for (auto volume : volumes) {

    for (double epsrel : epsrels) {
      constexpr int ndim = 8;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      czero_time_and_call<F_5_8D_alt,
                        ndim,
                        use_custom,
                        debug,
                        num_runs>("f5_alt", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 7;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      czero_time_and_call<F_5_7D_alt,
                        ndim,
                        use_custom,
                        debug,
                        num_runs>("f5_alt", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 6;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      czero_time_and_call<F_5_6D_alt,
                        ndim,
                        use_custom,
                        debug,
                        num_runs>("f5_alt", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 5;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      czero_time_and_call<F_5_5D_alt,
                        ndim,
                        use_custom,
                        debug,
                        num_runs>("f5_alt", epsrel, outfile, vol);
    }
  }

  outfile.close();
  return 0;
}
