#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"
#include "common/cuda/integrands.cuh"

template <typename F,
          int ndim,
          bool use_custom = false,
          int debug = 0,
          int runs_per_esprel = 10>
void
product_peak_time_and_call(std::string id,
                           double epsrel,
                           std::ostream& outfile,
                           quad::Volume<double, ndim>& vol)
{

  std::vector<double> peak_prominence = {40., 45., 50., 55., 60., 65., 75.};

  for (auto difficulty : peak_prominence) {
    using MilliSeconds =
      std::chrono::duration<double, std::chrono::milliseconds::period>;
    double constexpr epsabs = 1.0e-40;
    bool relerr_classification = true;
    Workspace<double, ndim, use_custom> workspace;
    F integrand;
    integrand.alpha = difficulty;
    integrand.set_true_value();
    auto print_custom = [=](bool use_custom_flag) {
      std::string to_print = use_custom_flag == true ? "custom" : "library";
      return to_print;
    };

    for (int i = 0; i < runs_per_esprel; i++) {
      auto const t0 = std::chrono::high_resolution_clock::now();
      size_t partitions_per_axis = get_partitions_per_axis(ndim);
      Sub_regions<double, ndim> sub_regions(partitions_per_axis);
      constexpr bool predict_split = false;
      constexpr bool collect_iters = false;

      numint::integration_result result =
        workspace.template integrate<F, predict_split, collect_iters, debug>(
          integrand, sub_regions, epsrel, epsabs, vol, relerr_classification);
      MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
      double const absolute_error =
        std::abs(result.estimate - integrand.true_value);

      std::cout.precision(17);
      /*if (i != 0)*/ {
        std::cout << id << "," << ndim << "," << vol.lows[0] << ","
                  << vol.highs[0] << "," << print_custom(use_custom) << ","
                  << difficulty << "," << std::fixed << std::scientific
                  << integrand.true_value << "," << epsrel << "," << epsabs
                  << "," << result.estimate << "," << result.errorest << ","
                  << result.nregions << "," << result.status << ","
                  << dt.count() << std::endl;

        outfile << id << "," << ndim << "," << std::scientific << vol.lows[0]
                << "," << vol.highs[0] << "," << print_custom(use_custom) << ","
                << difficulty << "," << integrand.true_value << "," << epsrel
                << "," << epsabs << "," << result.estimate << ","
                << result.errorest << "," << result.nregions << ","
                << result.iters << "," << result.status << "," << dt.count()
                << std::endl;
      }
    }
  }
}

int
main()
{
  constexpr bool use_custom = false;
  constexpr int debug = 0;
  constexpr int num_runs = 10;
  std::vector<double> epsrels = {
    1.e-3, 1.e-4, 1.e-5, 1.e-6, 1.e-7, 1.e-8, 1.e-9};
  std::vector<std::pair<double, double>> volumes = {{0, 1}};
  std::ofstream outfile("cuda_pagani_product_peaks.csv");
  outfile << "id, ndim, low, high, use_custom, peak_prominence, true_value, "
             "epsrel, epsabs, "
             "estimate, errorest, nregions, completed_iters, status, time"
          << std::endl;
  for (auto volume : volumes) {

    for (double epsrel : epsrels) {
      constexpr int ndim = 8;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      product_peak_time_and_call<F_2_8D_alt, ndim, use_custom, debug, num_runs>(
        "F_2_alt", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 7;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      product_peak_time_and_call<F_2_7D_alt, ndim, use_custom, debug, num_runs>(
        "F_2_alt", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 6;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      product_peak_time_and_call<F_2_6D_alt, ndim, use_custom, debug, num_runs>(
        "F_2_alt", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 5;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      product_peak_time_and_call<F_2_5D_alt, ndim, use_custom, debug, num_runs>(
        "F_2_alt", epsrel, outfile, vol);
    }
  }

  outfile.close();
  return 0;
}
