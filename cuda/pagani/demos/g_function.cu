#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"
#include "common/cuda/integrands.cuh"

template <typename F,
          int ndim,
          bool use_custom = false,
          int debug = 0,
          int runs_per_esprel = 10>
bool
g_func_time_and_call(std::string id,
                     double epsrel,
                     std::ostream& outfile,
                     quad::Volume<double, ndim>& vol)
{

  using MilliSeconds =
    std::chrono::duration<double, std::chrono::milliseconds::period>;
  double constexpr epsabs = 1.0e-40;
  bool good = false;
  bool relerr_classification = true;
  Workspace<double, ndim, debug, use_custom> workspace;
  F integrand;
  integrand.set_true_value(vol.lows[0], vol.highs[0]);
  auto print_custom = [=](bool use_custom_flag) {
    std::string to_print = use_custom_flag == true ? "custom" : "library";
    return to_print;
  };

  for (int i = 0; i < runs_per_esprel; i++) {
    auto const t0 = std::chrono::high_resolution_clock::now();
    size_t partitions_per_axis = get_partitions_per_axis(ndim);
    Sub_regions<double, ndim> sub_regions(partitions_per_axis);
    constexpr bool predict_split = false;
    constexpr bool collect_iters = false;

    numint::integration_result result =
      workspace.template integrate<F, predict_split, collect_iters>(
        integrand, sub_regions, epsrel, epsabs, vol, relerr_classification);
    MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
    double const absolute_error =
      std::abs(result.estimate - integrand.true_value);

    if (result.status == 0) {
      good = true;
    }

    std::cout.precision(17);
    /*if (i != 0)*/ {
      std::cout << id << "," << ndim << "," << vol.lows[0] << ","
                << vol.highs[0] << "," << print_custom(use_custom) << ","
                << std::fixed << std::scientific << integrand.true_value << ","
                << epsrel << "," << epsabs << "," << result.estimate << ","
                << result.errorest << "," << result.nregions << ","
                << result.status << "," << dt.count() << std::endl;

      outfile << id << "," << ndim << "," << std::scientific << vol.lows[0]
              << "," << vol.highs[0] << "," << print_custom(use_custom) << ","
              << integrand.true_value << "," << epsrel << "," << epsabs << ","
              << result.estimate << "," << result.errorest << ","
              << result.nregions << "," << result.iters << "," << result.status
              << "," << dt.count() << std::endl;
    }
  }
  return good;
}

int
main()
{
  constexpr bool use_custom = false;
  constexpr int debug = 0;
  constexpr int num_runs = 10;
  std::vector<double> epsrels = {
    1.e-3, 1.e-4, 1.e-5, 1.e-6, 1.e-7, 1.e-8, 1.e-9};
  std::vector<std::pair<double, double>> volumes = {{0, 1}, {0, 2}, {0, 3}};
  std::ofstream outfile("cuda_pagani_g_func.csv");
  outfile << "id, ndim, low, high, use_custom, true_value, epsrel, epsabs, "
             "estimate, errorest, nregions, completed_iters, status, time"
          << std::endl;

  for (auto volume : volumes) {

    for (double epsrel : epsrels) {
      constexpr int ndim = 10;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      g_func_time_and_call<G_func_10D, ndim, use_custom, debug, num_runs>(
        "G_func", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 9;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      g_func_time_and_call<G_func_9D, ndim, use_custom, debug, num_runs>(
        "G_func", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 8;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      g_func_time_and_call<G_func_8D, ndim, use_custom, debug, num_runs>(
        "G_func", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 7;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      g_func_time_and_call<G_func_7D, ndim, use_custom, debug, num_runs>(
        "G_func", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 6;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      g_func_time_and_call<G_func_6D, ndim, use_custom, debug, num_runs>(
        "G_func", epsrel, outfile, vol);
    }

    for (double epsrel : epsrels) {
      constexpr int ndim = 5;
      quad::Volume<double, ndim> vol(volume.first, volume.second);
      g_func_time_and_call<G_func_5D, ndim, use_custom, debug, num_runs>(
        "G_func", epsrel, outfile, vol);
    }
  }

  outfile.close();
  return 0;
}
