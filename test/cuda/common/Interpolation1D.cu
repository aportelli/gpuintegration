#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include "catch2/catch.hpp"
#include "common/cuda/Interp1D.cuh"
#include "common/cuda/cudaMemoryUtil.h"

#include <array>
#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>

__global__ void
Evaluate(quad::Interp1D interpolator,
         size_t size,
         double* input,
         double* results)
{
  for (size_t i = 0; i < size; i++) {
    results[i] = interpolator(input[i]);
  }
}

__global__ void
Evaluate(quad::Interp1D interpolator, double value, double* result)
{
  *result = interpolator(value);
}

void
interpolate_at_knots()
{
  const size_t s = 9;
  std::array<double, s> xs = {1., 2., 3., 4., 5., 6, 7., 8., 9.};
  std::array<double, s> ys = xs;

  auto Transform = [](std::array<double, s>& ys) {
    for (double& elem : ys)
      elem = 2 * elem * (3 - elem) * std::cos(elem);
  };

  Transform(ys);
  quad::Interp1D interpObj(xs, ys);

  double* input = quad::cuda_malloc_managed<double>(s);
  for (size_t i = 0; i < s; i++)
    input[i] = xs[i];

  double* results = quad::cuda_malloc_managed<double>(s);

  Evaluate<<<1, 1>>>(interpObj, s, input, results);
  hipDeviceSynchronize();

  for (std::size_t i = 0; i < s; ++i) {
    CHECK(ys[i] == results[i]);
  }
  hipFree(results);
  hipFree(input);
}

void
interpolate_on_quadratic()
{
  const size_t s = 5;
  std::array<double, s> xs = {1., 2., 3., 4., 5.};
  std::array<double, s> ys = xs;

  auto Transform = [](std::array<double, s>& ys) {
    for (auto& elem : ys)
      elem = elem * elem;
  };
  Transform(ys);
  quad::Interp1D interpObj(xs, ys);

  double* result = quad::cuda_malloc_managed<double>(1);
  double interp_point = 1.41421;
  double true_interp_res = 2.24263;
  Evaluate<<<1, 1>>>(interpObj, interp_point, result);
  hipDeviceSynchronize();
  CHECK(*result == Approx(true_interp_res).epsilon(1e-4));

  interp_point = 2.41421;
  true_interp_res = 6.07105;
  Evaluate<<<1, 1>>>(interpObj, interp_point, result);
  hipDeviceSynchronize();
  CHECK(*result == Approx(true_interp_res).epsilon(1e-4));

  interp_point = 3.41421;
  true_interp_res = 11.89947;
  Evaluate<<<1, 1>>>(interpObj, interp_point, result);
  hipDeviceSynchronize();
  CHECK(*result == Approx(true_interp_res).epsilon(1e-4));

  interp_point = 4.41421;
  true_interp_res = 19.72789;
  Evaluate<<<1, 1>>>(interpObj, interp_point, result);
  hipDeviceSynchronize();
  CHECK(*result == Approx(true_interp_res).epsilon(1e-4));

  hipFree(result);
}

TEST_CASE("Interp1D exact at knots", "[interpolation][1d]")
{
  interpolate_at_knots();
}

TEST_CASE()
{
  interpolate_on_quadratic();
}
